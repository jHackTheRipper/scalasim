#include "hip/hip_runtime.h"

#include <iostream>
#include <list>
#include <vector>
#include <utility>
#include <iterator>
#include <functional>
#include <boost/iterator/zip_iterator.hpp>
#include <cstdlib>

#include "zip.h"
#include "algorithm.h" // std::my_copy_if

#include <hip/hip_runtime.h>
#include "cuda_utils/cuda_util.h"

// ------ Schelling -------

/** Definitions concerning the place type */
typedef std::pair<int, int> Position;
typedef int Place;
typedef std::vector<Position>  	  PositionList;
typedef std::vector<PositionList> PositionMatrix;
typedef std::vector<std::vector<Place> >  PlaceMatrix;

/** Constants from the simulation model */
namespace Schelling {
    const Place Free  = 0;
    const Place White = 1;
    const Place Black = 2;

    const int side = 500;

    const double freeP = 0.02;
    const double whiteP = 0.5;
}

const float nbThreads = 256.0;

class State {
public:
	int side_;
	PlaceMatrix matrix_;

	// CUDA members
	int* flatPosTab_device;
	int* flatPosTab_host;
	int* movingTab_device;
	int* movingTab_host;

private:
	inline int pmod(int i, int j) const {
		int m = i %j;
		if (m < 0) 	return m + j;
		return m;
	}

	static void randomCell (Place& inPlace) {
	    if(drand48() < Schelling::freeP) inPlace = Schelling::Free;
	    else if(drand48() < Schelling::whiteP) inPlace = Schelling::White;
	    else inPlace = Schelling::Black;
	}

public:
	explicit State(int inSize) :side_(inSize), matrix_(side_) {
        // reserve space for matrix columns
        // must use std::bind2nd -> http://www.cplusplus.com/reference/functional/mem_fun_ref/?kw=mem_fun_ref [Return Value]
        std::for_each (matrix_.begin(), matrix_.end(), std::bind2nd(std::mem_fun_ref(&std::vector<Place>::reserve), side_));
	}

    void init() {
        srand48(0);

        PlaceMatrix::iterator rows = matrix_.begin();
        PlaceMatrix::iterator rowsEnd = matrix_.end();

        // randomly initialize each cell according to model parameters
        for (; rows != rowsEnd; ++rows) {
            std::for_each(rows->begin(), rows->end(), randomCell);
        }

        // CUDA init
        flatPosTab_host = new int[side_ * side_];
        cutilSafeCall( hipMalloc(&flatPosTab_device, side_ * side_ * sizeof(int)) );

        movingTab_host = new int[side_ * side_];
        cutilSafeCall( hipMalloc(&movingTab_device, side_ * side_ * sizeof(int)) );
    }

	Place& operator() (int i, int j) {
		return matrix_[pmod(i, side_)][pmod(j, side_)];
	}
	const Place& operator() (int i, int j) const {
		return matrix_[pmod(i, side_)][pmod(j, side_)];
	}

};

// --- helpers ---
bool isFree(const Place& inPlace) {
    return Schelling::Free == inPlace;
}

int position2Int(const State& inState, const Position& inPosition) {
	return inState(inPosition.first, inPosition.second);
}

PositionList moving(const State& inState, float inSimilarWanted) {

	PlaceMatrix::const_iterator matrixBegin = inState.matrix_.begin();
	PlaceMatrix::const_iterator matrixEnd = inState.matrix_.end();

	for (; matrixBegin != matrixEnd; ++matrixBegin) {
		// determine current row in the matrix
		int offset = (matrixEnd - matrixBegin) * inState.side_;
		// then copy the Int value of its Position of the current row
		std::copy(matrixBegin->begin(), matrixBegin->end(), inState.flatPosTab_host + offset);
	}

	cutilSafeCall( hipMemcpy(inState.flatPosTab_device, inState.flatPosTab_host, inState.side_ * inState.side_ * sizeof(int), hipMemcpyHostToDevice ) );
	long int nbBlocks = static_cast<int>( ceil (inState.side_ * inState.side_ / nbThreads ) );
//	movingKernel <<< nbBlocks, nbThreads >>>(inState.flatPosTab_device, inState.movingTab_device, inState.side_ * inState.side_);
	cutilSafeCall( hipMemcpy(inState.movingTab_host, inState.movingTab_device, inState.side_ * inState.side_ * sizeof(int), hipMemcpyDeviceToHost ) );

	PositionList moving;

	for (int i = 0; i < inState.side_; ++i) {
		for (int j = 0; i < inState.side_; ++j) {
			if (inState.movingTab_host[i * inState.side_ + j] < inSimilarWanted) 	moving.push_back(std::make_pair<int, int>(i, j));
		}
	}

	return moving;
}


PositionList freeCells(const State& inState) {
    PositionList freeCells;

    PlaceMatrix::const_iterator first = inState.matrix_.begin();
    PlaceMatrix::const_iterator end = inState.matrix_.end();
    PositionList::iterator current = freeCells.begin();

    for (; first != end; ++first) {
        current = std::my_copy_if(first->begin(), first->end(), first, end, current, isFree);
    }

    return freeCells;
}

/** Functor moving agents in the next state, and freeing their previous Place */
struct CopyMoves {

	const State& currentState;
	State& 		 nextState;

	explicit CopyMoves (const State& inCurrentState, State& inNextState)
		:currentState(inCurrentState), nextState(inNextState)
	{}

	void operator() (const boost::tuple<Position, Position>& inTuple) {
		nextState(inTuple.get<0>().first, inTuple.get<0>().second) = currentState(inTuple.get<1>().first, inTuple.get<1>().second);
        nextState(inTuple.get<1>().first, inTuple.get<1>().second) = Schelling::Free;
     }
};

State step(const State& inState) {
	PositionList wantToMove  = moving(inState, 0.65);
	PositionList free 		 = freeCells(inState);

	std::random_shuffle(wantToMove.begin(), wantToMove.end());
	std::random_shuffle(free.begin(), free.end());

	zip_container<PositionList, PositionList> moves = zip(wantToMove, free);
	State nextState(inState);

	CopyMoves functor(inState, nextState);

	for_each(moves.begin(), moves.end(), functor);

	return nextState;
}

void simulation(State& inoutState, int nbSteps) {

	for (int i = nbSteps; 0 != i; --i) {
		std::cout << i << " steps left" << std::endl;
		step(inoutState);
	}

	std::cout << "done" << std::endl;
}


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	State initialState(Schelling::side);
	initialState.init();
	simulation( initialState, 500);

	return 0;
}
